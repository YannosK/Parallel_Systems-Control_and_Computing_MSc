#include "hip/hip_runtime.h"
#include "files.h"
#include "timer.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#define SOFTENING 1e-9f
#define DEBUG

/*
 * Each body contains x, y, and z coordinate positions,
 * as well as velocities in the x, y, and z directions.
 */
typedef struct {
    float x, y, z, vx, vy, vz;
} Body;

/*
 * Kernel to calculate the gravitational impact
 * of all bodies in the system on all others.
 *
 * @param p pointer to a buffer that holds all the bodies
 * @param dt minimum time differential
 * @param n the total number of bodies
 */
__global__ void bodyForce(Body *p, float dt, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for(int i = index; i < n; i += stride) {
        float Fx = 0.0f;
        float Fy = 0.0f;
        float Fz = 0.0f;

        for(int j = 0; j < n; j++) {
            float dx = p[j].x - p[i].x;
            float dy = p[j].y - p[i].y;
            float dz = p[j].z - p[i].z;
            float distSqr = dx * dx + dy * dy + dz * dz + SOFTENING;
            float invDist = rsqrtf(distSqr);
            float invDist3 = invDist * invDist * invDist;

            Fx += dx * invDist3;
            Fy += dy * invDist3;
            Fz += dz * invDist3;
        }

        p[i].vx += dt * Fx;
        p[i].vy += dt * Fy;
        p[i].vz += dt * Fz;
    }
}

/**
 * Kernel to integrate positions of bodies.
 * This should be done after `bodyForce`.
 *
 * @param p pointer to a buffer that holds all the bodies
 * @param dt minimum time differential
 * @param n the total number of bodies
 */
__global__ void integratePosition(Body *p, float dt, int n) {
    size_t index = threadIdx.x + blockIdx.x * blockDim.x;
    size_t stride = blockDim.x * gridDim.x;

    for(size_t i = index; i < n; i += stride) {
        p[i].x += p[i].vx * dt;
        p[i].y += p[i].vy * dt;
        p[i].z += p[i].vz * dt;
    }
}

int main(const int argc, const char **argv) {
    /**------------------------------------------------------------------------------------
     * Command line arguments
     ------------------------------------------------------------------------------------*/

    // The assessment will test against both 2<11 and 2<15.
    // Feel free to pass the command line argument 15 when you generate ./nbody
    // report files
    int nBodies = 2 << 11;
    if(argc > 1)
        nBodies = 2 << atoi(argv[1]);

#ifdef DEBUG
    printf("\nProblem size: %d\n", nBodies);
#endif

    /**------------------------------------------------------------------------------------
     * Provided assesments - DO NOT MODIFY
     ------------------------------------------------------------------------------------*/

    // The assessment will pass hidden initialized values to check for
    // correctness. You should not make changes to these files, or else the
    // assessment will not work.
    const char *initialized_values;
    const char *solution_values;

    if(nBodies == 2 << 11) {
        initialized_values = "09-nbody/files/initialized_4096";
        solution_values = "09-nbody/files/solution_4096";
    } else { // nBodies == 2<<15
        initialized_values = "09-nbody/files/initialized_65536";
        solution_values = "09-nbody/files/solution_65536";
    }

    if(argc > 2)
        initialized_values = argv[2];
    if(argc > 3)
        solution_values = argv[3];

    /**------------------------------------------------------------------------------------
     * GPU parameters update
     ------------------------------------------------------------------------------------*/

    hipDeviceProp_t props;

    int deviceId;
    int warpSize;
    int maxThreadsPerBlock;
    size_t threadsPerBlock;
    size_t numberOfBlocks;

    hipGetDevice(&deviceId);
    hipGetDeviceProperties(&props, deviceId);

    warpSize = props.warpSize;
    maxThreadsPerBlock = props.maxThreadsPerBlock;

#ifdef DEBUG
    printf("Warp size: %d\n", warpSize);
    printf("Max threads per block: %d\n", maxThreadsPerBlock);
#endif

    threadsPerBlock = 4 * warpSize;
    if(threadsPerBlock > maxThreadsPerBlock)
        threadsPerBlock /= 4;

    if(nBodies % threadsPerBlock != 0)
        numberOfBlocks = (nBodies / threadsPerBlock) + 1;
    else
        numberOfBlocks = (nBodies / threadsPerBlock);

#ifdef DEBUG
    printf("\nNumber of blocks: %lu\n", numberOfBlocks);
    printf("Threads per block: %lu\n", threadsPerBlock);
#endif

    /**------------------------------------------------------------------------------------
     * Parameters initialization
     ------------------------------------------------------------------------------------*/

    hipError_t allocErr, bodyForceErr, integrationErr, asyncErr, fetchErr;

    const float dt = 0.01f; // Time step
    const int nIters = 10;  // Simulation iterations

    int bytes = nBodies * sizeof(Body); // total size of all the bodie in memory
    float *buf; // buffer that holds all the data bodies in memory - element
                // scope access

    allocErr = hipMallocManaged(&buf, bytes);
    if(allocErr != hipSuccess) {
        printf(
            "Error in allocation of the bodies buffer: %s\n",
            hipGetErrorString(allocErr)
        );
        return 1;
    }
    Body *p = (Body *)buf; // buffer that holds all the bodies in memory - body
                           // struct scope access

    // initialization of starting positions and velocities
    read_values_from_file(initialized_values, buf, bytes);

    fetchErr = hipMemPrefetchAsync(p, bytes, deviceId);
    if(fetchErr != hipSuccess) {
        printf(
            "Error in unififed memory pre-fetching HtoD: %s\n",
            hipGetErrorString(fetchErr)
        );
        return 5;
    }

    double totalTime = 0.0;

    /**------------------------------------------------------------------------------------
     * Simulation of `nBodies`
     ------------------------------------------------------------------------------------*/

    /*
     * This simulation will run for `nIters` cycles of time, calculating
     * gravitational interaction amongst bodies, and adjusting their positions
     * to reflect.
     */

    for(int iter = 0; iter < nIters; iter++) {
        StartTimer();

        // compute interbody forces
        bodyForce<<<numberOfBlocks, threadsPerBlock>>>(p, dt, nBodies);
        bodyForceErr = hipGetLastError();

        // integrate positions based on calculated forces
        integratePosition<<<numberOfBlocks, threadsPerBlock>>>(p, dt, nBodies);
        integrationErr = hipGetLastError();

        asyncErr = hipDeviceSynchronize();

        const double tElapsed = GetTimer() / 1000.0;
        totalTime += tElapsed;

        if(asyncErr != hipSuccess) {
            printf(
                "Asynchronous error in iteration %d: %s\n", iter,
                hipGetErrorString(asyncErr)
            );
            return 4;
        }
        if(bodyForceErr != hipSuccess) {
            printf(
                "bodyForce kernel error in iteration %d: %s\n", iter,
                hipGetErrorString(bodyForceErr)
            );
            return 3;
        }
        if(integrationErr != hipSuccess) {
            printf(
                "integratePosition kernel error in iteration %d: %s\n", iter,
                hipGetErrorString(integrationErr)
            );
            return 3;
        }
    }

    /**------------------------------------------------------------------------------------
     * Execution results
     ------------------------------------------------------------------------------------*/

    fetchErr = hipMemPrefetchAsync(p, bytes, hipCpuDeviceId);
    if(fetchErr != hipSuccess) {
        printf(
            "Error in unififed memory pre-fetching DtoH: %s\n",
            hipGetErrorString(fetchErr)
        );
        return 5;
    }

    double avgTime = totalTime / (double)(nIters);
#ifdef DEBUG
    printf("\nAverage execution time: %lf\n", avgTime);
#endif

    float billionsOfOpsPerSecond = 1e-9 * nBodies * nBodies / avgTime;
    write_values_to_file(solution_values, buf, bytes);

    printf("%0.3f Billion Interactions / second\n", billionsOfOpsPerSecond);

    /**------------------------------------------------------------------------------------
     * Deallocation and termination
     ------------------------------------------------------------------------------------*/

    allocErr = hipFree(buf);
    if(allocErr != hipSuccess) {
        printf(
            "Error in de-allocation of the bodies buffer: %s\n",
            hipGetErrorString(allocErr)
        );
        return 2;
    }

    return 0;
}
